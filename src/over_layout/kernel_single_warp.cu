#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>
#include <math.h>
#include <vector>
#include "../common/param.h"
// #include "common.hpp"
// #include "stencil_part.cpp"

using namespace nvcuda;
using namespace std;

__global__ void mma_run(\
    half * A_Padding, float *S, float *C,\
    int offset_n, int offset_m,\
    half *matrix_row, \
    half *matrix_col,\
    int row_addr,\
    int col_addr, \
    int store_addr, \
    int row_pos_min , int row_pos_max, int col_pos_min, int col_pos_max,\
    int stencil_part_size, int *stencil_part_type,int *stencil_part_pos, int *stencil_part_order,\
    int N, int padding)//分块边长大于16是必然的
{

    extern __shared__ half data[];
    half *row_data = (half *)(data + row_addr);
    half *col_data = (half *)(data + col_addr);
    float *store_data = (float *)(data + store_addr);

    ////////////////////Block_N == stencil_row_size; Block_M == stencil_col_size
    int size = N + 2 * padding;
    int tid = threadIdx.x + threadIdx.y*blockDim.x;
    int offset = (padding + blockIdx.y*Core_M)*size + padding + blockIdx.x*Core_N;//当前线程块对应的数据位置因为横向和纵向填充都是padding，但分块其实不是方形

    //一次性加载行拆分部分的全部
    int row_offset = offset - offset_m*size - offset_n;//row_pos_min和col_pos_min是负数
    int times = (Block_N*Block_M - 1)/(blockDim.x*blockDim.y) + 1;
    #pragma unroll
    for(int i = 0; i < times; ++i)
    {
        if(i*blockDim.x*blockDim.y + tid < Block_N*Block_M)
        {
            (row_data)[i*blockDim.x*blockDim.y + tid] = \
            (A_Padding + row_offset)[(i*blockDim.x*blockDim.y + tid)%Block_N + (i*blockDim.x*blockDim.y + tid)/Block_N*size];    
        }
    }
    __syncthreads();

    // 对列拆分参数矩阵对应的网格进行加载，直接从刚才读取的行拆分部分进行转置，采取行读列存的形式，而且用不了伪向量化
    // 一次性加载列拆分部分的全部
    times = (Block_M*Block_N - 1)/(blockDim.x*blockDim.y) + 1;
    #pragma unroll
    for(int i = 0; i < times; ++i)
    {
        if(i*blockDim.x*blockDim.y + tid < Block_M*Block_N)
        {
            col_data[(i*blockDim.x*blockDim.y + tid)%Block_N*Block_M + (i*blockDim.x*blockDim.y + tid)/Block_N] = \
            row_data[i*blockDim.x*blockDim.y + tid];
        }
    }
    __syncthreads();

    // 调用tcu进行计算(注意本实现只有一个warp)
    #pragma unroll
    for(int m = 0; m < Block_M; m += WMMA_M)//总所周知，stencil被拆分成了多个部分，每个部分都是一个参数矩阵和对应的网格一一对应，这一共有stencil_part_size个对应
    {
        #pragma unroll
        for(int n = 0; n < Block_N; n += WMMA_N)//这里采取对分块中一个一个WMMA_M*WMMA_N的小块依次计算，每个小块需要将所有对应全部遍历一遍
        {
            wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
            wmma::fill_fragment(c_frag, 0.0f);
            #pragma unroll
            for(int i = 0; i < stencil_part_size; ++i)
            {
                if(stencil_part_type[i] == 0)//横向
                {
                    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;
                    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> b_frag;
                    int data_offset = (stencil_part_pos[i])*Block_N;//row_data
                    int stencil_offset = stencil_part_order[i]*Block_N*Block_N;//matrix_row
                    #pragma unroll
                    for(int k = 0; k < Block_N; k += WMMA_K)
                    {
                        wmma::load_matrix_sync(a_frag, row_data + data_offset + m*Block_N + k, Block_N);//网格
                        wmma::load_matrix_sync(b_frag, matrix_row + stencil_offset + k*Block_N + n, Block_N);//参数矩阵
                        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
                    }
                }
                else
                {
                    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;//参数矩阵
                    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;//网格 注意这里应该是col_major 我们构造的
                    int data_offset = (stencil_part_pos[i])*Block_M;//col_data
                    int stencil_offset = stencil_part_order[i]*Block_M*Block_M;//matrix_col
                    #pragma unroll
                    for(int k = 0; k < Block_M; k += WMMA_K)
                    {
                        wmma::load_matrix_sync(a_frag,matrix_col + stencil_offset + m*Block_M + k,Block_M);
                        wmma::load_matrix_sync(b_frag, col_data + data_offset + n*Block_M + k, Block_M);//stride怎么填，数据偏移怎么写，非常有说法
                        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
                    }
                }
            }
            wmma::store_matrix_sync(store_data + m*Block_N + n, c_frag, Block_N, wmma::mem_row_major);
        }
    }

    __syncthreads();

    //将结果写回
    int store_offset = offset_m*Block_N + offset_n;
    times = (Core_N*Core_M - 1)/(blockDim.x*blockDim.y) + 1;
    #pragma unroll
    for(int i = 0; i < times; ++i)
    {
        int pos_m = blockIdx.y*Core_M + (i*blockDim.x*blockDim.y + tid)/Core_N;
        int pos_n = blockIdx.x*Core_N + (i*blockDim.x*blockDim.y + tid)%Core_N;
        if(pos_m < N && pos_n < N && i*blockDim.x*blockDim.y + tid < Core_N*Core_M)
        {
            C[pos_m*N + pos_n] = \
            (store_data + store_offset)[(i*blockDim.x*blockDim.y + tid)%Core_N + (i*blockDim.x*blockDim.y + tid)/Core_N*Block_N];
        }
    }



    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\nrow_data:\n");
    //     printf("times:%d\n", times);
    //     for(int i = 0; i < (Block_M + row_pos_max - row_pos_min); ++i)
    //     {
    //         for(int j = 0; j < Block_N; ++j)
    //         {
    //             printf("%4.0f ", __half2float(row_data[i * Block_N + j]));
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();
    
    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\ncol_data:\n");
    //     printf("times:%d\n", times);
    //     for(int i = 0; i < Block_N + col_pos_max - col_pos_min; ++i)
    //     {
    //         for(int j = 0; j < Block_M; ++j)
    //         {
    //             printf("%4.0f ", __half2float(col_data[i * Block_M + j]));
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\nstore_data:\n");
    //     printf("times:%d\n", times);
    //     for(int i = 0; i < Block_M; ++i)
    //     {
    //         for(int j = 0; j < Block_N; ++j)
    //         {
    //             printf("%4.0f ", store_data[i * Block_N + j]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\nstore_data_part:\n");
    //     printf("times:%d\n", times);
    //     for(int i = offset_m; i < offset_m + Core_M; ++i)
    //     {
    //         for(int j = offset_n; j < offset_n + Core_N; ++j)
    //         {
    //             printf("%4.0f ", store_data[i * Block_N + j]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\nC_store_data:\n");
    //     printf("times:%d\n", times);
    //     for(int i = 0; i < Core_M; ++i)
    //     {
    //         for(int j = 0; j < Core_N; ++j)
    //         {
    //             printf("%4.0f ", (C + C_offset)[i * N + j]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

}