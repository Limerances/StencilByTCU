#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>
#include <math.h>
#include <vector>
#include "../common/param.h"
// #include "common.hpp"
// #include "stencil_part.cpp"

using namespace nvcuda;
using namespace std;

__global__ void mma_run(\
    half * A_Padding, float *S, float *C,\
    int offset_n, int offset_m,\
    half *matrix_col,\
    int col_addr, \
    int col_pos_min, int col_pos_max,\
    int stencil_part_size, int *stencil_part_type,int *stencil_part_pos, int *stencil_part_order,\
    int N, int padding)//分块边长大于16是必然的
{

    extern __shared__ half data[];
    half *col_data = (half *)(data + col_addr);
    __shared__ float store_data[Block_M*Block_N];

    ////////////////////Block_N == stencil_row_size; Block_M == stencil_col_size
    int size = N + 2 * padding;
    int tid = threadIdx.x + threadIdx.y*blockDim.x;
    int offset = (padding + blockIdx.y*Core_M)*size + padding + blockIdx.x*Core_N;//当前线程块对应的数据位置因为横向和纵向填充都是padding，但分块其实不是方形

    // 对列拆分参数矩阵对应的网格进行加载，采取行读列存的形式，而且用不了伪向量化
    // 一次性加载列拆分部分的全部
    int col_offset = offset - offset_m*size - offset_n + col_pos_min;//row_pos_min和col_pos_min是负数
    int Block_N_expand = Block_N + col_pos_max - col_pos_min;
    int times = (Block_M*Block_N_expand - 1)/(blockDim.x*blockDim.y) + 1;
    // #pragma unroll
    // for(int i = 0; i < times; ++i)
    // {
    //     if(i*blockDim.x*blockDim.y + tid < Block_M*Block_N_expand)
    //     {
    //         data[(i*blockDim.x*blockDim.y + tid)%Block_N_expand*Block_M + (i*blockDim.x*blockDim.y + tid)/Block_N_expand] = \
    //         (A_Padding + col_offset)[(i*blockDim.x*blockDim.y + tid)%Block_N_expand + (i*blockDim.x*blockDim.y + tid)/Block_N_expand*size];
    //     }
    // }
    __syncthreads();

    // 调用tcu进行计算(注意本实现只有一个warp)
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
    wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;//参数矩阵
    wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;//网格 注意这里应该是col_major 我们构造的
    wmma::fill_fragment(c_frag, 0.0f);
    #pragma unroll
    for(int i = 0; i < stencil_part_size; ++i)
    {
        int data_offset = (stencil_part_pos[i])*Block_M;//col_data
        int stencil_offset = stencil_part_order[i]*Block_M*Block_M;//matrix_col

        wmma::load_matrix_sync(a_frag,matrix_col + stencil_offset,Block_M);
        wmma::load_matrix_sync(b_frag, col_data + data_offset, Block_M);//stride怎么填，数据偏移怎么写，非常有说法
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }
    wmma::store_matrix_sync(store_data, c_frag, Block_N, wmma::mem_row_major);

    __syncthreads();

    //将结果写回
    int store_offset = offset_m*Block_N + offset_n;
    times = (Core_N*Core_M - 1)/(blockDim.x*blockDim.y) + 1;
    #pragma unroll
    for(int i = 0; i < times; ++i)
    {
        int pos_m = blockIdx.y*Core_M + (i*blockDim.x*blockDim.y + tid)/Core_N;
        int pos_n = blockIdx.x*Core_N + (i*blockDim.x*blockDim.y + tid)%Core_N;
        if(pos_m < N && pos_n < N && i*blockDim.x*blockDim.y + tid < Core_N*Core_M)
        {
            C[pos_m*N + pos_n] = \
            (store_data + store_offset)[(i*blockDim.x*blockDim.y + tid)%Core_N + (i*blockDim.x*blockDim.y + tid)/Core_N*Block_N];
        }
    }



    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\nrow_data:\n");
    //     printf("times:%d\n", times);
    //     for(int i = 0; i < (Block_M + row_pos_max - row_pos_min); ++i)
    //     {
    //         for(int j = 0; j < Block_N; ++j)
    //         {
    //             printf("%4.0f ", __half2float(row_data[i * Block_N + j]));
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();
    
    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\ncol_data:\n");
    //     printf("times:%d\n", times);
    //     for(int i = 0; i < Block_N + col_pos_max - col_pos_min; ++i)
    //     {
    //         for(int j = 0; j < Block_M; ++j)
    //         {
    //             printf("%4.0f ", __half2float(col_data[i * Block_M + j]));
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\nstore_data:\n");
    //     printf("times:%d\n", times);
    //     for(int i = 0; i < Block_M; ++i)
    //     {
    //         for(int j = 0; j < Block_N; ++j)
    //         {
    //             printf("%4.0f ", store_data[i * Block_N + j]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\nstore_data_part:\n");
    //     printf("times:%d\n", times);
    //     for(int i = offset_m; i < offset_m + Core_M; ++i)
    //     {
    //         for(int j = offset_n; j < offset_n + Core_N; ++j)
    //         {
    //             printf("%4.0f ", store_data[i * Block_N + j]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\nC_store_data:\n");
    //     printf("times:%d\n", times);
    //     for(int i = 0; i < Core_M; ++i)
    //     {
    //         for(int j = 0; j < Core_N; ++j)
    //         {
    //             printf("%4.0f ", (C + C_offset)[i * N + j]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

}