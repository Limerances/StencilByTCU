#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <mma.h>
#include <stdio.h>
#include <math.h>
#include <vector>
#include "../common/param.h"
// #include "common.hpp"
// #include "stencil_part.cpp"

using namespace nvcuda;
using namespace std;

__global__ void mma_run(\
    half * A_Padding, float *S, float *C,\
    half *matrix_col,\
    int halo_addr,\
    int S_addr,\
    int col_pos_min, int col_pos_max,\
    int stencil_part_size, int *stencil_part_type,int *stencil_part_pos, int *stencil_part_order,\
    int N, int padding)//分块边长大于16是必然的
{

    extern __shared__ half data[];
    
    half *col_data = (half *)data;
    half *halo_data = (half *)(data + halo_addr);
    float *S_data = (float *)(data + S_addr);
    float *store_data = (float *)(data);

    ////////////////////Block_N == stencil_row_size; Block_M == stencil_col_size
    int size = N + 2 * padding;
    int tid = threadIdx.x + threadIdx.y*blockDim.x;
    int offset = (padding + blockIdx.y*Block_M)*size + padding + blockIdx.x*Block_N;//当前线程块对应的数据位置因为横向和纵向填充都是padding，但分块其实不是方形

    //对列拆分参数矩阵对应的网格进行加载(这里需要一个转置)，采取行读列存的形式，而且用不了伪向量化
    int col_offset = offset + col_pos_min;//col_pos_min是负数
    int Block_N_expand = Block_N + col_pos_max - col_pos_min;//用于在A_Padding中定位，要取的数据一行连续的长度为Block_N_expand
    int times = (Block_M*(Block_N + col_pos_max - col_pos_min) - 1)/(blockDim.x*blockDim.y) + 1;
    #pragma unroll
    for(int i = 0; i < times; ++i)
    {
        if(i*blockDim.x*blockDim.y + tid < Block_M*Block_N_expand)
        {
            col_data[(i*blockDim.x*blockDim.y + tid)%Block_N_expand*Block_M + (i*blockDim.x*blockDim.y + tid)/Block_N_expand] = \
            A_Padding[col_offset + (i*blockDim.x*blockDim.y + tid)%Block_N_expand + (i*blockDim.x*blockDim.y + tid)/Block_N_expand*size];
        }
    }

    // //加载参数矩阵
    times = (stencil_shape_M*stencil_shape_N - 1)/(blockDim.x*blockDim.y) + 1;
    #pragma unroll
    for(int i = 0; i < times; ++i)
    {
        if(i*blockDim.x*blockDim.y + tid < stencil_shape_M*stencil_shape_N)
        {
            S_data[(i*blockDim.x*blockDim.y + tid)/stencil_shape_N + (i*blockDim.x*blockDim.y + tid)%stencil_shape_N*stencil_shape_M] = \
            S[i*blockDim.x*blockDim.y + tid];
        }
    }


    __syncthreads();

    //调用tcu进行计算(注意本实现只有一个warp)(部分区域的计算)
    wmma::fragment<wmma::accumulator, WMMA_M, WMMA_N, WMMA_K, float> c_frag;
    wmma::fill_fragment(c_frag, 0.0f);
    #pragma unroll
    for(int i = 0; i < stencil_part_size; ++i)
    {
        wmma::fragment<wmma::matrix_a, WMMA_M, WMMA_N, WMMA_K, half, wmma::row_major> a_frag;//参数矩阵
        wmma::fragment<wmma::matrix_b, WMMA_M, WMMA_N, WMMA_K, half, wmma::col_major> b_frag;//网格 注意这里应该是col_major 我们构造的
        int data_offset = (stencil_part_pos[i] + abs(col_pos_min))*Block_M;//col_data
        int stencil_offset = stencil_part_order[i]*Block_M*Block_M;//matrix_col

        wmma::load_matrix_sync(a_frag,matrix_col + stencil_offset,Block_M);
        wmma::load_matrix_sync(b_frag, col_data + data_offset, Block_M);//stride怎么填，数据偏移怎么写，非常有说法
        wmma::mma_sync(c_frag, a_frag, b_frag, c_frag);
    }
    wmma::store_matrix_sync(store_data , c_frag, Block_N, wmma::mem_row_major);

    __syncthreads();

    // halo区,上下左右分别为halo1234区
    int halo1_m = stencil_core_M;
    int halo1_n = Block_N + stencil_shape_N - 1;
    half *halo1 = (half *)(halo_data);
    int halo2_m = stencil_shape_M - stencil_core_M - 1;
    int halo2_n = Block_N + stencil_shape_N - 1;
    half *halo2 = (half *)(halo1 + halo1_m*halo1_n);
    int halo3_m = Block_M + stencil_shape_M - 1;
    int halo3_n = stencil_core_N;
    half *halo3 = (half *)(halo2 + halo2_m*halo2_n);
    int halo4_m = Block_M + stencil_shape_M - 1;
    int halo4_n = stencil_shape_N - stencil_core_N - 1;
    half *halo4 = (half *)(halo3 + halo3_m*halo3_n);

    // 加载halo区数据
    // halo1
    int halo_offset = offset - halo1_m*size - halo3_n;
    times = (halo1_m*halo1_n - 1)/(blockDim.x*blockDim.y) + 1;
    #pragma unroll
    for(int i = 0; i < times; ++i)
    {
        if(i*blockDim.x*blockDim.y + tid < halo1_m*halo1_n)
        {
            halo1[i*blockDim.x*blockDim.y + tid] = A_Padding[halo_offset + (i*blockDim.x*blockDim.y + tid)%halo1_n + (i*blockDim.x*blockDim.y + tid)/halo1_n*size];
        }
    }

    //halo2
    halo_offset = offset + Block_M*size - halo3_n;
    times = (halo2_m*halo2_n - 1)/(blockDim.x*blockDim.y) + 1;
    #pragma unroll
    for(int i = 0; i < times; ++i)
    {
        if(i*blockDim.x*blockDim.y + tid < halo2_m*halo2_n)
        {
            halo2[i*blockDim.x*blockDim.y + tid] = A_Padding[halo_offset + (i*blockDim.x*blockDim.y + tid)%halo2_n + (i*blockDim.x*blockDim.y + tid)/halo2_n*size];
        }
    }

    __syncthreads();
    //计算halo区
    
    if(tid < 16)
    {
        #pragma unroll
        for(int i = 0; i < stencil_part_size; ++i)
        {
            int s_index = (stencil_core_N + stencil_part_pos[i])*stencil_shape_M;
            #pragma unroll
            for(int row = 0; row < stencil_core_M; ++row)//halo1
            {
                #pragma unroll
                for(int num_per_row = 0; num_per_row <= row; ++num_per_row)
                {
                    // int a = -1;
                    store_data[num_per_row*Block_N + tid] += \
                    __half2float(halo1[stencil_core_N + tid + stencil_part_pos[i] + row*halo1_n]);
                    // S_data[s_index + row - num_per_row];
                }
            }
            #pragma unroll
            for(int row = 0; row < stencil_shape_M - stencil_core_M - 1; ++row)//halo2
            {
                #pragma unroll
                for(int num_per_row = 0; num_per_row <= row; ++num_per_row)
                {
                    // int a = 1;
                    store_data[(Block_M - 1 - num_per_row)*Block_N + tid] += \
                    __half2float(halo2[stencil_core_N + tid + stencil_part_pos[i] + (halo2_m - 1 - row)*halo2_n]);
                    // S_data[s_index + stencil_shape_M - 1 - row + num_per_row];  
                }
            }
        }

    }
    

    __syncthreads();


    //将结果写回
    int C_offset = blockIdx.y*Block_M*N + blockIdx.x*Block_N;
    int Block_N_float4 = Block_N/4;
    int N_float4 = N/4;
    times = (Block_N_float4*Block_M - 1)/(blockDim.x*blockDim.y) + 1;
    #pragma unroll
    for(int i = 0; i < times; ++i)
    {
        if(i*blockDim.x*blockDim.y + tid < Block_N_float4*Block_M)
        {
            ((float4 *)(C + C_offset))[(i*blockDim.x*blockDim.y + tid)%Block_N_float4 + (i*blockDim.x*blockDim.y + tid)/Block_N_float4*N_float4] = \
            ((float4 *)(store_data))[i*blockDim.x*blockDim.y + tid];
        }
    }


    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\nrow_data:\n");
    //     printf("times:%d\n", times);
    //     for(int i = 0; i < (Block_M + row_pos_max - row_pos_min); ++i)
    //     {
    //         for(int j = 0; j < Block_N; ++j)
    //         {
    //             printf("%4.0f ", __half2float(row_data[i * Block_N + j]));
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\ncol_data:\n");
    //     printf("times:%d\n", times);
    //     for(int i = 0; i < Block_N + col_pos_max - col_pos_min; ++i)
    //     {
    //         for(int j = 0; j < Block_M; ++j)
    //         {
    //             printf("%4.0f ", __half2float(col_data[i * Block_M + j]));
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\nstore_data:\n");
    //     printf("times:%d\n", times);
    //     for(int i = 0; i < Block_M; ++i)
    //     {
    //         for(int j = 0; j < Block_N; ++j)
    //         {
    //             printf("%4.0f ", store_data[i * Block_N + j]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\nC_store_data:\n");
    //     printf("times:%d\n", times);
    //     for(int i = 0; i < Block_M; ++i)
    //     {
    //         for(int j = 0; j < Block_N; ++j)
    //         {
    //             printf("%4.0f ", (C + C_offset)[i * N + j]);
    //         }
    //         printf("\n");
    //     }
    //     printf("\n");
    // }
    // __syncthreads();

    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\nhalo1:\n");
    //     for(int i = 0; i < halo1_m; ++i)
    //     {
    //         for(int j = 0; j < halo1_n; ++j)
    //         {
    //             printf("%4.0f ", __half2float(halo1[i * halo1_n + j]));
    //         }
    //         printf("\n");
    //     }
    //     printf("\nhalo2:\n");
    //     for(int i = 0; i < halo2_m; ++i)
    //     {
    //         for(int j = 0; j < halo2_n; ++j)
    //         {
    //             printf("%4.0f ", __half2float(halo2[i * halo2_n + j]));
    //         }
    //         printf("\n");
    //     }
    //     printf("\nhalo3:\n");
    //     for(int i = 0; i < halo3_m; ++i)
    //     {
    //         for(int j = 0; j < halo3_n; ++j)
    //         {
    //             printf("%4.0f ", __half2float(halo3[i * halo3_n + j]));
    //         }
    //         printf("\n");
    //     }
    //     printf("\nhalo4:\n");
    //     for(int i = 0; i < halo4_m; ++i)
    //     {
    //         for(int j = 0; j < halo4_n; ++j)
    //         {
    //             printf("%4.0f ", __half2float(halo4[i * halo4_n + j]));
    //         }
    //         printf("\n");
    //     }
    // }
    // __syncthreads();

    // __syncthreads();
    // if(threadIdx.x == 0 && threadIdx.y == 0 && blockIdx.x == 0 && blockIdx.y == 0)
    // {
    //     printf("\nS_data:\n");
    //     for(int i = 0; i < stencil_shape_M; ++i)
    //     {
    //         for(int j = 0; j < stencil_shape_N; ++j)
    //         {
    //             printf("%4.0f ", S_data[i * stencil_shape_N + j]);
    //         }
    //         printf("\n");
    //     }
    // }
    // __syncthreads();

}